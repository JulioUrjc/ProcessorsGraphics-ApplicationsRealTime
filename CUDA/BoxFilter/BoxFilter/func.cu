#include "hip/hip_runtime.h"
//****************************************************************************
// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//****************************************************************************

#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

#define BLOCK_SIZE 32

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess) {
		std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		exit(1);
	}
}

// Optimize for pointer aliasing using __restrict__ allows CUDA commpiler to use the read-only data cache and improves performance
__global__
void box_filter(const unsigned char* const inputChannel, unsigned char* const outputChannel, int numRows, int numCols, const float* __restrict__ filter, const int filterWidth)
{
	// NOTA: Que un thread tenga una posici�n correcta en 2D no quiere decir que al aplicar el filtro
	// los valores de sus vecinos sean correctos, ya que pueden salirse de la imagen.

	const unsigned int x = threadIdx.x;
	const unsigned int y = threadIdx.y;

	/// which thread is this?
	const int tx = blockIdx.x*blockDim.x + x;
	const int ty = blockIdx.y*blockDim.y + y;

	/// if thread is out the image
	if (tx >= numCols || ty >= numRows)
		return;

	const int filterRadius = filterWidth / 2;
	float value = 0.0f;

	/// Share memory
	__shared__ float ds_inputChannel[BLOCK_SIZE][BLOCK_SIZE];

	ds_inputChannel[y][x] = inputChannel[ty*numCols + tx];
	// case1: upper left
	//x = tx - filterRadius;
	//y = ty - filterRadius;
	//if (x < 0 || y < 0)
	//	ds_inputChannel[threadIdx.y][threadIdx.x] = 0;
	//else
	//	ds_inputChannel[threadIdx.y][threadIdx.x] = inputChannel[ty*numCols + tx - filterRadius - numCols*filterRadius];

	//// case2: upper right
	//x = tx + filterRadius;
	//y = ty - filterRadius;
	//if (x > numCols - 1 || y < 0)
	//	ds_inputChannel[threadIdx.y][threadIdx.x + blockDim.x] = 0;
	//else
	//	ds_inputChannel[threadIdx.y][threadIdx.x + blockDim.x] = inputChannel[ty*numCols + tx + filterRadius - numCols*filterRadius];

	//// case3: lower left
	//x = tx - filterRadius;
	//y = ty + filterRadius;
	//if (x < 0 || y > numRows - 1)
	//	ds_inputChannel[threadIdx.y + blockDim.y][threadIdx.x] = 0;
	//else
	//	ds_inputChannel[threadIdx.y + blockDim.y][threadIdx.x] = inputChannel[ty*numCols + tx - filterRadius + numCols*filterRadius];

	//// case4: lower right
	//x = tx + filterRadius;
	//y = ty + filterRadius;
	//if (x > numCols - 1 || y > numRows - 1)
	//	ds_inputChannel[threadIdx.y + blockDim.y][threadIdx.x + blockDim.x] = 0;
	//else
	//	ds_inputChannel[threadIdx.y + blockDim.y][threadIdx.x + blockDim.x] = inputChannel[ty*numCols + tx + filterRadius - numCols*filterRadius];

	__syncthreads();

	for (int i = 0; i < filterWidth; ++i){
		/// which pixel is?
		//int fx = blockIdx.x*blockDim.x + (threadIdx.x + i - filterRadius);
		int fx = blockIdx.x*blockDim.x + (threadIdx.x + i - filterRadius);
		int fsx = x + i - filterRadius;
		/// Clamp of neighbourds values
		if (fx < 0)  fx = 0;
		if (fx > numCols - 1)  fx = numCols - 1;

		for (int j = 0; j < filterWidth; ++j){
			/// which pixel is?
			int fy = blockIdx.y*blockDim.y + (threadIdx.y + j - filterRadius);
			int fsy = y + j - filterRadius;
			/// Clamp of neighbourds values
			if (fy < 0)  fy = 0;
			if (fy > numRows - 1)  fy = numRows - 1;
			/// Compute the value at the pixel and add it.
			if ((fsx >= 0) && (fsy >= 0) && (fsx <= BLOCK_SIZE - 1) && (fsy <= BLOCK_SIZE - 1))
				value += filter[j*filterWidth + i] * ds_inputChannel[fsy][fsx];
			else
				value += filter[j*filterWidth + i] * inputChannel[fy*numCols + fx];
		}
	}

	/// Whitout share memory
	//for (int i = 0; i < filterWidth; ++i){
	//	/// which pixel is?
	//	int fx = blockIdx.x*blockDim.x + (threadIdx.x + i - filterRadius);
	//	/// Clamp of neighbourds values
	//	if (fx < 0)  fx = 0;
	//	if (fx > numCols - 1)  fx = numCols - 1;

	//	for (int j = 0; j < filterWidth; ++j){
	//		/// which pixel is?
	//		int fy = blockIdx.y*blockDim.y + (threadIdx.y + j - filterRadius);
	//		/// Clamp of neighbourds values
	//		if (fy < 0)  fy = 0;
	//		if (fy > numRows - 1)  fy = numRows - 1;
	//		/// Compute the value at the pixel and add it.
	//		value += filter[j*filterWidth + i] * inputChannel[fy*numCols + fx];
	//	}
	//}

	/// Save the value at the outputChanel
	outputChannel[ty*numCols + tx] = value;
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
int numRows,
int numCols,
unsigned char* const redChannel,
unsigned char* const greenChannel,
unsigned char* const blueChannel)
{
	/// which thread is this?
	const int tx = blockIdx.x*blockDim.x + threadIdx.x;
	const int ty = blockIdx.y*blockDim.y + threadIdx.y;

	/// if thread is out the image
	if (tx >= numCols || ty >= numRows)
		return;

	/// Index to get the array position
	const int index = ty * numCols + tx;

	/// split colors
	redChannel[index] = inputImageRGBA[index].x;
	greenChannel[index] = inputImageRGBA[index].y;
	blueChannel[index] = inputImageRGBA[index].z;

}

//This kernel takes in three color channels and recombines them
//into one image. The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
const unsigned char* const greenChannel,
const unsigned char* const blueChannel,
uchar4* const outputImageRGBA,
int numRows,
int numCols)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	//make sure we don't try and access memory outside the image
	//by having any threads mapped there return early
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	unsigned char red = redChannel[thread_1D_pos];
	unsigned char green = greenChannel[thread_1D_pos];
	unsigned char blue = blueChannel[thread_1D_pos];

	//Alpha should be 255 for no transparency
	uchar4 outputPixel = make_uchar4(red, green, blue, 255);

	outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
//float         *d_filter;
__constant__ float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
	const float* const h_filter, const size_t filterWidth)
{

	//allocate memory for the three different channels
	checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRowsImage * numColsImage));
	checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
	checkCudaErrors(hipMalloc(&d_blue, sizeof(unsigned char) * numRowsImage * numColsImage));

	//Reservar memoria para el filtro en GPU: d_filter, la cual ya esta declarada
	// Copiar el filtro  (h_filter) a memoria global de la GPU (d_filter)
	checkCudaErrors(hipMalloc(&d_filter, sizeof(float)*filterWidth*filterWidth));
	checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float)*filterWidth*filterWidth, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_filter), h_filter, sizeof(float)*filterWidth*filterWidth));
}


void create_filter(float **h_filter, int *filterWidth){

	const int KernelWidth = 5; //OJO CON EL TAMA�O DEL FILTRO//
	//const int KernelWidth = 3;
	*filterWidth = KernelWidth;

	//create and fill the filter we will convolve with
	*h_filter = new float[KernelWidth * KernelWidth];

	/*
	//Filtro gaussiano: blur
	const float KernelSigma = 2.;

	float filterSum = 0.f; //for normalization

	for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
	for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
	float filterValue = expf( -(float)(c * c + r * r) / (2.f * KernelSigma * KernelSigma));
	(*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] = filterValue;
	filterSum += filterValue;
	}
	}

	float normalizationFactor = 1.f / filterSum;

	for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
	for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
	(*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] *= normalizationFactor;
	}
	}
	*/

	//Laplaciano 5x5
	(*h_filter)[0] = 0;   (*h_filter)[1] = 0;    (*h_filter)[2] = -1.;  (*h_filter)[3] = 0;    (*h_filter)[4] = 0;
	(*h_filter)[5] = 1.;  (*h_filter)[6] = -1.;  (*h_filter)[7] = -2.;  (*h_filter)[8] = -1.;  (*h_filter)[9] = 0;
	(*h_filter)[10] = -1.; (*h_filter)[11] = -2.; (*h_filter)[12] = 17.; (*h_filter)[13] = -2.; (*h_filter)[14] = -1.;
	(*h_filter)[15] = 1.; (*h_filter)[16] = -1.; (*h_filter)[17] = -2.; (*h_filter)[18] = -1.; (*h_filter)[19] = 0;
	(*h_filter)[20] = 0;  (*h_filter)[21] = 0;   (*h_filter)[22] = -1.; (*h_filter)[23] = 0;   (*h_filter)[24] = 0;

	//Crear los filtros segun necesidad
	//NOTA: cuidado al establecer el tama�o del filtro a utilizar

	//////// FILTROS DE NITIDEZ
	///Nitidez 5x5 - kernelWidth = 5
	//(*h_filter)[0] = 0;		(*h_filter)[1] = -1.;	(*h_filter)[2] = -1.;  (*h_filter)[3] = -1.;	(*h_filter)[4] = 0;
	//(*h_filter)[5] = -1.;	(*h_filter)[6] = 2.;	(*h_filter)[7] = -4.;  (*h_filter)[8] = 2.;		(*h_filter)[9] = -1.;
	//(*h_filter)[10] = -1.;	(*h_filter)[11] = -4.;	(*h_filter)[12] = 13.; (*h_filter)[13] = -4.;	(*h_filter)[14] = -1.;
	//(*h_filter)[15] = -1.;	(*h_filter)[16] = 2.;	(*h_filter)[17] = -4.; (*h_filter)[18] = 2.;	(*h_filter)[19] = -1.;
	//(*h_filter)[20] = 0;	(*h_filter)[21] = -1.;  (*h_filter)[22] = -1.; (*h_filter)[23] = -1.;   (*h_filter)[24] = 0;

	/// Nitidez 3x3
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = -1.;  
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 9.;	(*h_filter)[5] = -1.;	
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = -1.;	(*h_filter)[8] = -1.;

	/// Aumentar nitidez
	//(*h_filter)[0] = 0.;	(*h_filter)[1] = -0.25;	(*h_filter)[2] = 0.;
	//(*h_filter)[3] = -0.25;	(*h_filter)[4] = 2.;	(*h_filter)[5] = -0.25;
	//(*h_filter)[6] = 0.;	(*h_filter)[7] = -0.25;	(*h_filter)[8] = 0.;

	/// Aumentar nitidez 2
	//(*h_filter)[0] = -0.25;	(*h_filter)[1] = -0.25;	(*h_filter)[2] = -0.25;
	//(*h_filter)[3] = -0.25;	(*h_filter)[4] = 3.;	(*h_filter)[5] = -0.25;
	//(*h_filter)[6] = -0.25;	(*h_filter)[7] = -0.25;	(*h_filter)[8] = -0.25;

	//////// FILTROS DE GRADIENTE
	/// Gradiente este
	//(*h_filter)[0] = 1.;	(*h_filter)[1] = 0.;	(*h_filter)[2] = 1.;
	//(*h_filter)[3] = 2.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = -2.;
	//(*h_filter)[6] = 1.;	(*h_filter)[7] = 0.;	(*h_filter)[8] = -1.;

	/// Gradiente norte
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = -2.;	(*h_filter)[2] = -1.;
	//(*h_filter)[3] = 0.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = 0.;
	//(*h_filter)[6] = 1.;	(*h_filter)[7] = 2.;	(*h_filter)[8] = 1.;

	/// Gradiente nordeste
	//(*h_filter)[0] = 0.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = -2.;
	//(*h_filter)[3] = 1.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = -1.;
	//(*h_filter)[6] = 2.;	(*h_filter)[7] = 1.;	(*h_filter)[8] = 0.;

	/// Gradiente noroeste
	//(*h_filter)[0] = -2.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = 0.;
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = 1.;
	//(*h_filter)[6] = 0.;	(*h_filter)[7] = 1.;	(*h_filter)[8] = 2.;

	/// Gradiente sur
	//(*h_filter)[0] = 1.;	(*h_filter)[1] = 2.;	(*h_filter)[2] = 1.;
	//(*h_filter)[3] = 0.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = 0.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = -2.;	(*h_filter)[8] = -1.;

	/// Gradiente oeste
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = 0.;	(*h_filter)[2] = 1.;
	//(*h_filter)[3] = -2.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = 2.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = 0.;	(*h_filter)[8] = 1.;

	//////// FILTROS DE DETECCION DE LINEA
	/// Deteccion de linea horizontal
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = -1.;
	//(*h_filter)[3] = 2.;	(*h_filter)[4] = 2.;	(*h_filter)[5] = 2.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = -1.;	(*h_filter)[8] = -1.;

	/// Deteccion de linea diagonal izquierda
	//(*h_filter)[0] = 2.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = -1.;
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 2.;	(*h_filter)[5] = -1.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = -1.;	(*h_filter)[8] = 2.;

	/// Deteccion de linea diagonal derecha
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = 2.;
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 2.;	(*h_filter)[5] = -1.;
	//(*h_filter)[6] = 2.;	(*h_filter)[7] = -1.;	(*h_filter)[8] = -1.;

	/// Deteccion de linea vertical
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = 2.;	(*h_filter)[2] = -1.;
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 2.;	(*h_filter)[5] = -1.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = 2.;	(*h_filter)[8] = -1.;

	//////// FILTROS DE SUAVIZADO
	/// Media Aritmetica Suave - estandar 3x3
	//(*h_filter)[0] = 0.111;	(*h_filter)[1] = 0.111;	(*h_filter)[2] = 0.111;
	//(*h_filter)[3] = 0.111;	(*h_filter)[4] = 0.111;	(*h_filter)[5] = 0.111;
	//(*h_filter)[6] = 0.111;	(*h_filter)[7] = 0.111;	(*h_filter)[8] = 0.111;
	/// Media Aritmetica Suave - generica NxN
	//float value = 1/(float)(KernelWidth*KernelWidth);
	//for (int i = 0; i < KernelWidth; ++i)
	//	for (int j = 0; j < KernelWidth; ++j)
	//		(*h_filter)[i + KernelWidth * j] = value;

	/// Suavizado 3x3
	//(*h_filter)[0] = 1.;	(*h_filter)[1] = 2.;	(*h_filter)[2] = 1.;
	//(*h_filter)[3] = 2.;	(*h_filter)[4] = 4.;	(*h_filter)[5] = 2.;
	//(*h_filter)[6] = 1.;	(*h_filter)[7] = 2.;	(*h_filter)[8] = 1.;

	/// Suavizado 5x5
	//(*h_filter)[0] = 1.;	(*h_filter)[1] = 1.;    (*h_filter)[2] = 1.;	(*h_filter)[3] = 1.;    (*h_filter)[4] = 1.;
	//(*h_filter)[5] = 1.;	(*h_filter)[6] = 4.;	(*h_filter)[7] = 4.;	(*h_filter)[8] = 4.;	(*h_filter)[9] = 1.;
	//(*h_filter)[10] = 1.;	(*h_filter)[11] = 4.;	(*h_filter)[12] = 12.;	(*h_filter)[13] = 4.;	(*h_filter)[14] = 1.;
	//(*h_filter)[15] = 1.;	(*h_filter)[16] = 4.;	(*h_filter)[17] = 4.;	(*h_filter)[18] = 4.;	(*h_filter)[19] = 1.;
	//(*h_filter)[20] = 1.;	(*h_filter)[21] = 1.;   (*h_filter)[22] = 1.;	(*h_filter)[23] = 1.;   (*h_filter)[24] = 1.;

	//////// FILTROS DE SUAVIZADO
	/// Media Aritmetica Suave
	//(*h_filter)[0] = 0.111;	(*h_filter)[1] = 0.111;	(*h_filter)[2] = 0.111;
	//(*h_filter)[3] = 0.111;	(*h_filter)[4] = 0.111;	(*h_filter)[5] = 0.111;
	//(*h_filter)[6] = 0.111;	(*h_filter)[7] = 0.111;	(*h_filter)[8] = 0.111;
 

}


void convolution(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
	uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
	unsigned char *d_redFiltered,
	unsigned char *d_greenFiltered,
	unsigned char *d_blueFiltered,
	const int filterWidth)
{
	/// Calcular tama�os de bloque
	//	La tarjeta tiene un maximo de 1024 threads por bloque, por tanto el tamanyo de bloque es de 32
	const dim3 gridSize((numCols-1) / BLOCK_SIZE+1, (numRows-1) / BLOCK_SIZE+1, 1);
	const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);

	/// Lanzar kernel para separar imagenes RGBA en diferentes colores
	separateChannels << <gridSize, blockSize >> >(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);

	//Ejecutar convoluci�n. Una por canal
	box_filter << <gridSize, blockSize >> >(d_red, d_redFiltered, numRows, numCols, d_filter, filterWidth);
	box_filter << <gridSize, blockSize >> >(d_green, d_greenFiltered, numRows, numCols, d_filter, filterWidth);
	box_filter << <gridSize, blockSize >> >(d_blue, d_blueFiltered, numRows, numCols, d_filter, filterWidth);

	// Recombining the results. 
	recombineChannels << <gridSize, blockSize >> >(d_redFiltered,
		d_greenFiltered,
		d_blueFiltered,
		d_outputImageRGBA,
		numRows,
		numCols);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
//make sure you free any arrays that you allocated
void cleanup() {
	checkCudaErrors(hipFree(d_red));
	checkCudaErrors(hipFree(d_green));
	checkCudaErrors(hipFree(d_blue));
	checkCudaErrors(hipFree(d_filter));
}
