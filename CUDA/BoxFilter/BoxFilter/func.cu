#include "hip/hip_runtime.h"
//****************************************************************************
// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//****************************************************************************

#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

#define BLOCK_SIZE 32
#define FILTER_WIDTH 5
#define clamp(x,a,b) (__min(__max((x), a), b))

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess) {
		std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		exit(1);
	}
}

// Optimize for pointer aliasing using __restrict__ allows CUDA commpiler to use the read-only data cache and improves performance
__global__
void box_filter(const unsigned char* const inputChannel, unsigned char* const outputChannel, int numRows, int numCols, const float* __restrict__ filter, const int filterWidth)
//void box_filter(const unsigned char* const inputChannel, unsigned char* const outputChannel, int numRows, int numCols, const float* filter, const int filterWidth)
{
	// NOTA: Que un thread tenga una posici�n correcta en 2D no quiere decir que al aplicar el filtro
	// los valores de sus vecinos sean correctos, ya que pueden salirse de la imagen.

	const unsigned int x0 = threadIdx.x;
	const unsigned int y0 = threadIdx.y;

	/// which thread is this?
	const int tx = blockIdx.x*blockDim.x + x0;
	const int ty = blockIdx.y*blockDim.y + y0;

	/// if thread is out the image
	if (tx >= numCols || ty >= numRows)
		return;

	const int filterRadius = filterWidth / 2;
	float value = 0.0f;
	
	/// Share memory - Input tiles need to be larger than output tiles
	__shared__ float ds_inputChannel[BLOCK_SIZE+FILTER_WIDTH-1][BLOCK_SIZE+FILTER_WIDTH-1];

	// Each thread copy the vertex of the filter, the 4 corners - error some pixels
	int fx, fy; // Index for the filter corners
	// case1: upper left
	fx = tx - filterRadius;
	fy = ty - filterRadius;
	fx = clamp(fx, 0, numCols-1);
	fy = clamp(fy, 0, numRows-1);
	//baux = !((fx < 0) || (fy < 0));
	ds_inputChannel[x0][y0] = inputChannel[fy*numCols + fx];

	// case2: upper right
	fx = tx + filterRadius;
	fy = ty - filterRadius;
	fx = clamp(fx, 0, numCols - 1);
	fy = clamp(fy, 0, numRows - 1);
	//baux = !((fx > numCols - 1) || (fy < 0));
	ds_inputChannel[x0 + FILTER_WIDTH][y0] = inputChannel[fy*numCols + fx];

	// case3: lower left
	fx = tx - filterRadius;
	fy = ty + filterRadius;
	fx = clamp(fx, 0, numCols - 1);
	fy = clamp(fy, 0, numRows - 1);
	//baux = !((fx < 0) || (fy > numRows - 1));
	ds_inputChannel[x0][y0 + FILTER_WIDTH] = inputChannel[fy*numCols + fx];

	// case4: lower right
	fx = tx + filterRadius;
	fy = ty + filterRadius;
	fx = clamp(fx, 0, numCols - 1);
	fy = clamp(fy, 0, numRows - 1);
	//baux = !((fx > numCols - 1) || (fy > numRows - 1));
	ds_inputChannel[x0 + FILTER_WIDTH][y0 + FILTER_WIDTH] = inputChannel[fy*numCols + fx];

	__syncthreads(); // SyncThreads to have all the share memory complete

	for (int i = 0; i < FILTER_WIDTH; ++i){
		for (int j = 0; j < FILTER_WIDTH; ++j){
			value += filter[j*FILTER_WIDTH + i] * ds_inputChannel[x0+i][y0+j];
		}
	}

	/// Whitout share memory
	/*for (int i = 0; i < filterWidth; ++i){
		/// which pixel is?
		int fx = blockIdx.x*blockDim.x + (threadIdx.x + i - filterRadius);
		/// Clamp of neighbourds values
		if (fx < 0)  fx = 0;
		if (fx > numCols - 1)  fx = numCols - 1;

		for (int j = 0; j < filterWidth; ++j){
			/// which pixel is?
			int fy = blockIdx.y*blockDim.y + (threadIdx.y + j - filterRadius);
			/// Clamp of neighbourds values
			if (fy < 0)  fy = 0;
			if (fy > numRows - 1)  fy = numRows - 1;
			/// Compute the value at the pixel and add it.
			value += filter[j*filterWidth + i] * inputChannel[fy*numCols + fx];
		}
	}*/


	/// Save the value at the outputChanel
	outputChannel[ty*numCols + tx] = value;
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
int numRows,
int numCols,
unsigned char* const redChannel,
unsigned char* const greenChannel,
unsigned char* const blueChannel)
{
	/// which thread is this?
	const int tx = blockIdx.x*blockDim.x + threadIdx.x;
	const int ty = blockIdx.y*blockDim.y + threadIdx.y;

	/// if thread is out the image
	if (tx >= numCols || ty >= numRows)
		return;

	/// Index to get the array position
	const int index = ty * numCols + tx;

	/// split colors
	redChannel[index] = inputImageRGBA[index].x;
	greenChannel[index] = inputImageRGBA[index].y;
	blueChannel[index] = inputImageRGBA[index].z;

}

//This kernel takes in three color channels and recombines them
//into one image. The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
const unsigned char* const greenChannel,
const unsigned char* const blueChannel,
uchar4* const outputImageRGBA,
int numRows,
int numCols)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	//make sure we don't try and access memory outside the image
	//by having any threads mapped there return early
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	unsigned char red = redChannel[thread_1D_pos];
	unsigned char green = greenChannel[thread_1D_pos];
	unsigned char blue = blueChannel[thread_1D_pos];

	//Alpha should be 255 for no transparency
	uchar4 outputPixel = make_uchar4(red, green, blue, 255);

	outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
//float         *d_filter;
//__constant__ float d_filter[FILTER_WIDTH*FILTER_WIDTH];
__constant__ float *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
	const float* const h_filter, const size_t filterWidth)
{

	//allocate memory for the three different channels
	checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRowsImage * numColsImage));
	checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
	checkCudaErrors(hipMalloc(&d_blue, sizeof(unsigned char) * numRowsImage * numColsImage));

	//Reservar memoria para el filtro en GPU: d_filter, la cual ya esta declarada
	// Copiar el filtro  (h_filter) a memoria global de la GPU (d_filter)
	checkCudaErrors(hipMalloc(&d_filter, sizeof(float)*filterWidth*filterWidth));
	checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float)*filterWidth*filterWidth, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_filter), h_filter, sizeof(float)*filterWidth*filterWidth));
}


void create_filter(float **h_filter, int *filterWidth){

	const int KernelWidth = 5; //OJO CON EL TAMA�O DEL FILTRO//
	//const int KernelWidth = 3;
	*filterWidth = KernelWidth;

	//create and fill the filter we will convolve with
	*h_filter = new float[KernelWidth * KernelWidth];

	/*
	//Filtro gaussiano: blur
	const float KernelSigma = 2.;

	float filterSum = 0.f; //for normalization

	for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
	for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
	float filterValue = expf( -(float)(c * c + r * r) / (2.f * KernelSigma * KernelSigma));
	(*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] = filterValue;
	filterSum += filterValue;
	}
	}

	float normalizationFactor = 1.f / filterSum;

	for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
	for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
	(*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] *= normalizationFactor;
	}
	}
	*/

	//The same image
	//(*h_filter)[0] = 0;		(*h_filter)[1] = 0;		(*h_filter)[2] = 0;		(*h_filter)[3] = 0;		(*h_filter)[4] = 0;		(*h_filter)[5] = 0;		(*h_filter)[6] = 0;		(*h_filter)[7] = 0;		(*h_filter)[8] = 0;		(*h_filter)[9] = 0;		(*h_filter)[10] = 0;
	//(*h_filter)[11] = 0;	(*h_filter)[12] = 0;	(*h_filter)[13] = 0;	(*h_filter)[14] = 0;	(*h_filter)[15] = 0;	(*h_filter)[16] = 0;	(*h_filter)[17] = 0;	(*h_filter)[18] = 0;	(*h_filter)[19] = 0;	(*h_filter)[20] = 0;	(*h_filter)[21] = 0;
	//(*h_filter)[22] = 0;	(*h_filter)[23] = 0;	(*h_filter)[24] = 0;	(*h_filter)[25] = 0;	(*h_filter)[26] = 0;	(*h_filter)[27] = 0;	(*h_filter)[28] = 0;	(*h_filter)[29] = 0;	(*h_filter)[30] = 0;	(*h_filter)[31] = 0;	(*h_filter)[32] = 0;
	//(*h_filter)[33] = 0;	(*h_filter)[34] = 0;	(*h_filter)[35] = 0;	(*h_filter)[36] = 0;	(*h_filter)[37] = 0;	(*h_filter)[38] = 0;	(*h_filter)[39] = 0;	(*h_filter)[40] = 0;	(*h_filter)[41] = 0;	(*h_filter)[42] = 0;	(*h_filter)[43] = 0;
	//(*h_filter)[44] = 0;	(*h_filter)[45] = 0;	(*h_filter)[46] = 0;	(*h_filter)[47] = 0;	(*h_filter)[48] = 0;	(*h_filter)[49] = 0;	(*h_filter)[50] = 0;	(*h_filter)[51] = 0;	(*h_filter)[52] = 0;	(*h_filter)[53] = 0;	(*h_filter)[54] = 0;
	//(*h_filter)[55] = 0;	(*h_filter)[56] = 0;	(*h_filter)[57] = 0;	(*h_filter)[58] = 0;	(*h_filter)[59] = 0;	(*h_filter)[60] = 1.;	(*h_filter)[61] = 0;	(*h_filter)[62] = 0;	(*h_filter)[63] = 0;	(*h_filter)[64] = 0;	(*h_filter)[65] = 0;
	//(*h_filter)[66] = 0;	(*h_filter)[67] = 0;	(*h_filter)[68] = 0;	(*h_filter)[69] = 0;	(*h_filter)[70] = 0;	(*h_filter)[71] = 0;	(*h_filter)[72] = 0;	(*h_filter)[73] = 0;	(*h_filter)[74] = 0;	(*h_filter)[75] = 0;	(*h_filter)[76] = 0;
	//(*h_filter)[77] = 0;	(*h_filter)[78] = 0;	(*h_filter)[79] = 0;	(*h_filter)[80] = 0;	(*h_filter)[81] = 0;	(*h_filter)[82] = 0;	(*h_filter)[83] = 0;	(*h_filter)[84] = 0;	(*h_filter)[85] = 0;	(*h_filter)[86] = 0;	(*h_filter)[87] = 0;
	//(*h_filter)[88] = 0;	(*h_filter)[89] = 0;	(*h_filter)[90] = 0;	(*h_filter)[91] = 0;	(*h_filter)[92] = 0;	(*h_filter)[93] = 0;	(*h_filter)[94] = 0;	(*h_filter)[95] = 0;	(*h_filter)[96] = 0;	(*h_filter)[97] = 0;	(*h_filter)[98] = 0;
	//(*h_filter)[99] = 0;	(*h_filter)[100] = 0;	(*h_filter)[101] = 0;	(*h_filter)[102] = 0;	(*h_filter)[103] = 0;	(*h_filter)[104] = 0;	(*h_filter)[105] = 0;	(*h_filter)[106] = 0;	(*h_filter)[107] = 0;	(*h_filter)[108] = 0;	(*h_filter)[109] = 0;
	//(*h_filter)[110] = 0;	(*h_filter)[111] = 0;	(*h_filter)[112] = 0;	(*h_filter)[113] = 0;	(*h_filter)[114] = 0;	(*h_filter)[115] = 0;	(*h_filter)[116] = 0;	(*h_filter)[117] = 0;	(*h_filter)[118] = 0;	(*h_filter)[119] = 0;	(*h_filter)[120] = 0;

	//Laplaciano 5x5
	(*h_filter)[0] = 0;   (*h_filter)[1] = 0;    (*h_filter)[2] = -1.;  (*h_filter)[3] = 0;    (*h_filter)[4] = 0;
	(*h_filter)[5] = 1.;  (*h_filter)[6] = -1.;  (*h_filter)[7] = -2.;  (*h_filter)[8] = -1.;  (*h_filter)[9] = 0;
	(*h_filter)[10] = -1.; (*h_filter)[11] = -2.; (*h_filter)[12] = 17.; (*h_filter)[13] = -2.; (*h_filter)[14] = -1.;
	(*h_filter)[15] = 1.; (*h_filter)[16] = -1.; (*h_filter)[17] = -2.; (*h_filter)[18] = -1.; (*h_filter)[19] = 0;
	(*h_filter)[20] = 0;  (*h_filter)[21] = 0;   (*h_filter)[22] = -1.; (*h_filter)[23] = 0;   (*h_filter)[24] = 0;

	//Crear los filtros segun necesidad
	//NOTA: cuidado al establecer el tama�o del filtro a utilizar

	//////// FILTROS DE NITIDEZ
	///Nitidez 5x5 - kernelWidth = 5
	//(*h_filter)[0] = 0;		(*h_filter)[1] = -1.;	(*h_filter)[2] = -1.;  (*h_filter)[3] = -1.;	(*h_filter)[4] = 0;
	//(*h_filter)[5] = -1.;	(*h_filter)[6] = 2.;	(*h_filter)[7] = -4.;  (*h_filter)[8] = 2.;		(*h_filter)[9] = -1.;
	//(*h_filter)[10] = -1.;	(*h_filter)[11] = -4.;	(*h_filter)[12] = 13.; (*h_filter)[13] = -4.;	(*h_filter)[14] = -1.;
	//(*h_filter)[15] = -1.;	(*h_filter)[16] = 2.;	(*h_filter)[17] = -4.; (*h_filter)[18] = 2.;	(*h_filter)[19] = -1.;
	//(*h_filter)[20] = 0;	(*h_filter)[21] = -1.;  (*h_filter)[22] = -1.; (*h_filter)[23] = -1.;   (*h_filter)[24] = 0;

	/// Nitidez 3x3
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = -1.;  
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 9.;	(*h_filter)[5] = -1.;	
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = -1.;	(*h_filter)[8] = -1.;

	/// Aumentar nitidez
	//(*h_filter)[0] = 0.;	(*h_filter)[1] = -0.25;	(*h_filter)[2] = 0.;
	//(*h_filter)[3] = -0.25;	(*h_filter)[4] = 2.;	(*h_filter)[5] = -0.25;
	//(*h_filter)[6] = 0.;	(*h_filter)[7] = -0.25;	(*h_filter)[8] = 0.;

	/// Aumentar nitidez 2
	//(*h_filter)[0] = -0.25;	(*h_filter)[1] = -0.25;	(*h_filter)[2] = -0.25;
	//(*h_filter)[3] = -0.25;	(*h_filter)[4] = 3.;	(*h_filter)[5] = -0.25;
	//(*h_filter)[6] = -0.25;	(*h_filter)[7] = -0.25;	(*h_filter)[8] = -0.25;

	//////// FILTROS DE GRADIENTE
	/// Gradiente este
	//(*h_filter)[0] = 1.;	(*h_filter)[1] = 0.;	(*h_filter)[2] = 1.;
	//(*h_filter)[3] = 2.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = -2.;
	//(*h_filter)[6] = 1.;	(*h_filter)[7] = 0.;	(*h_filter)[8] = -1.;

	/// Gradiente norte
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = -2.;	(*h_filter)[2] = -1.;
	//(*h_filter)[3] = 0.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = 0.;
	//(*h_filter)[6] = 1.;	(*h_filter)[7] = 2.;	(*h_filter)[8] = 1.;

	/// Gradiente nordeste
	//(*h_filter)[0] = 0.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = -2.;
	//(*h_filter)[3] = 1.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = -1.;
	//(*h_filter)[6] = 2.;	(*h_filter)[7] = 1.;	(*h_filter)[8] = 0.;

	/// Gradiente noroeste
	//(*h_filter)[0] = -2.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = 0.;
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = 1.;
	//(*h_filter)[6] = 0.;	(*h_filter)[7] = 1.;	(*h_filter)[8] = 2.;

	/// Gradiente sur
	//(*h_filter)[0] = 1.;	(*h_filter)[1] = 2.;	(*h_filter)[2] = 1.;
	//(*h_filter)[3] = 0.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = 0.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = -2.;	(*h_filter)[8] = -1.;

	/// Gradiente oeste
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = 0.;	(*h_filter)[2] = 1.;
	//(*h_filter)[3] = -2.;	(*h_filter)[4] = 0.;	(*h_filter)[5] = 2.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = 0.;	(*h_filter)[8] = 1.;

	//////// FILTROS DE DETECCION DE LINEA
	/// Deteccion de linea horizontal
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = -1.;
	//(*h_filter)[3] = 2.;	(*h_filter)[4] = 2.;	(*h_filter)[5] = 2.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = -1.;	(*h_filter)[8] = -1.;

	/// Deteccion de linea diagonal izquierda
	//(*h_filter)[0] = 2.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = -1.;
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 2.;	(*h_filter)[5] = -1.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = -1.;	(*h_filter)[8] = 2.;

	/// Deteccion de linea diagonal derecha
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = -1.;	(*h_filter)[2] = 2.;
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 2.;	(*h_filter)[5] = -1.;
	//(*h_filter)[6] = 2.;	(*h_filter)[7] = -1.;	(*h_filter)[8] = -1.;

	/// Deteccion de linea vertical
	//(*h_filter)[0] = -1.;	(*h_filter)[1] = 2.;	(*h_filter)[2] = -1.;
	//(*h_filter)[3] = -1.;	(*h_filter)[4] = 2.;	(*h_filter)[5] = -1.;
	//(*h_filter)[6] = -1.;	(*h_filter)[7] = 2.;	(*h_filter)[8] = -1.;

	//////// FILTROS DE SUAVIZADO
	/// Media Aritmetica Suave - estandar 3x3
	//(*h_filter)[0] = 0.111;	(*h_filter)[1] = 0.111;	(*h_filter)[2] = 0.111;
	//(*h_filter)[3] = 0.111;	(*h_filter)[4] = 0.111;	(*h_filter)[5] = 0.111;
	//(*h_filter)[6] = 0.111;	(*h_filter)[7] = 0.111;	(*h_filter)[8] = 0.111;
	/// Media Aritmetica Suave - generica NxN
	//float value = 1/(float)(KernelWidth*KernelWidth);
	//for (int i = 0; i < KernelWidth; ++i)
	//	for (int j = 0; j < KernelWidth; ++j)
	//		(*h_filter)[i + KernelWidth * j] = value;

	/// Suavizado 3x3
	//(*h_filter)[0] = 1.;	(*h_filter)[1] = 2.;	(*h_filter)[2] = 1.;
	//(*h_filter)[3] = 2.;	(*h_filter)[4] = 4.;	(*h_filter)[5] = 2.;
	//(*h_filter)[6] = 1.;	(*h_filter)[7] = 2.;	(*h_filter)[8] = 1.;

	/// Suavizado 5x5
	//(*h_filter)[0] = 1.;	(*h_filter)[1] = 1.;    (*h_filter)[2] = 1.;	(*h_filter)[3] = 1.;    (*h_filter)[4] = 1.;
	//(*h_filter)[5] = 1.;	(*h_filter)[6] = 4.;	(*h_filter)[7] = 4.;	(*h_filter)[8] = 4.;	(*h_filter)[9] = 1.;
	//(*h_filter)[10] = 1.;	(*h_filter)[11] = 4.;	(*h_filter)[12] = 12.;	(*h_filter)[13] = 4.;	(*h_filter)[14] = 1.;
	//(*h_filter)[15] = 1.;	(*h_filter)[16] = 4.;	(*h_filter)[17] = 4.;	(*h_filter)[18] = 4.;	(*h_filter)[19] = 1.;
	//(*h_filter)[20] = 1.;	(*h_filter)[21] = 1.;   (*h_filter)[22] = 1.;	(*h_filter)[23] = 1.;   (*h_filter)[24] = 1.;

	//////// FILTROS DE SUAVIZADO
	/// Media Aritmetica Suave
	//(*h_filter)[0] = 0.111;	(*h_filter)[1] = 0.111;	(*h_filter)[2] = 0.111;
	//(*h_filter)[3] = 0.111;	(*h_filter)[4] = 0.111;	(*h_filter)[5] = 0.111;
	//(*h_filter)[6] = 0.111;	(*h_filter)[7] = 0.111;	(*h_filter)[8] = 0.111;
 

}


void convolution(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
	uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
	unsigned char *d_redFiltered,
	unsigned char *d_greenFiltered,
	unsigned char *d_blueFiltered,
	const int filterWidth)
{
	/// Calcular tama�os de bloque
	//	La tarjeta tiene un maximo de 1024 threads por bloque, por tanto el tamanyo de bloque es de 32
	const dim3 gridSize((numCols-1) / BLOCK_SIZE+1, (numRows-1) / BLOCK_SIZE+1, 1);
	const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);

	/// Lanzar kernel para separar imagenes RGBA en diferentes colores
	separateChannels << <gridSize, blockSize >> >(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);
	hipDeviceSynchronize();

	//Ejecutar convoluci�n. Una por canal
	box_filter << <gridSize, blockSize >> >(d_red, d_redFiltered, numRows, numCols, d_filter, filterWidth);
	box_filter << <gridSize, blockSize >> >(d_green, d_greenFiltered, numRows, numCols, d_filter, filterWidth);
	box_filter << <gridSize, blockSize >> >(d_blue, d_blueFiltered, numRows, numCols, d_filter, filterWidth);
	hipDeviceSynchronize();

	// Recombining the results. 
	recombineChannels << <gridSize, blockSize >> >(d_redFiltered, d_greenFiltered, d_blueFiltered, d_outputImageRGBA, numRows, numCols);

	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
//make sure you free any arrays that you allocated
void cleanup() {
	checkCudaErrors(hipFree(d_red));
	checkCudaErrors(hipFree(d_green));
	checkCudaErrors(hipFree(d_blue));
	checkCudaErrors(hipFree(d_filter));
}
