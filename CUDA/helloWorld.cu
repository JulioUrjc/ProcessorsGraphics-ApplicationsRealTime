#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 16
#define BLOCK_WIDTH 16 // num threads by block

__global__ void kernel(char * m){

	printf("%s\n",m);
   return;
}

int main (int argc, char ** argv) {


   int tamM = 8;
   char * m  = (char *) malloc(tamM);
   char * dm = NULL;

   _snprintf_s(m,8,8, "Hello!!\0");

   fprintf(stderr,"Al hacer la reserva: %s \n",hipGetErrorString(hipMalloc(&dm,tamM)));
   fprintf(stderr,"Al hacer la copia:   %s \n",hipGetErrorString(hipMemcpy(dm,m,tamM,hipMemcpyHostToDevice)));
   fprintf(stderr,"Soy el HOST voy a mandar a dispositivo: %s\n",m);

   //Launch kernel
   kernel<<<NUM_THREADS/BLOCK_WIDTH,BLOCK_WIDTH>>>(dm);

   fprintf(stderr,"Mensaje: %s\n",m);

   hipFree(dm);
   
   return 0;
}

